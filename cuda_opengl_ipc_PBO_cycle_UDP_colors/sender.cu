#include "hip/hip_runtime.h"
#include <QCoreApplication>
#include <QUdpSocket>
#include <QTimer>
#include <QHostAddress>

#include "global_include.h"

#define CX -0.7436438870371587f
#define CY +0.13182590420531197f

__global__ void fill_matrix_linear_rgba(unsigned char* data, int width, int height, int frame) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x < width && y < height) {
        int idx = (y * width + x) * 4; // 4 canali
        data[idx + 0] = (x + frame) % 256; // R
        data[idx + 1] = (y + frame) % 256; // G
        data[idx + 2] = ((x + y) + frame) % 256; // B
        data[idx + 3] = 255; // A
    }
}

__global__ void fill_matrix_spiral_rgba(unsigned char* data, int width, int height, int frame) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x < width && y < height) {
        int idx = (y * width + x) * 4;

        float cx = width * 0.5f;
        float cy = height * 0.5f;
        float dx = x - cx;
        float dy = y - cy;
        float r = sqrtf(dx*dx + dy*dy);
        float theta = atan2f(dy, dx);
        float t = frame * 0.2f;
        // float lin_spiral = sinf(r * 0.05f + theta - t);
        float log_spiral = sinf(10.0f * logf(r + 1e-6f) + theta - t);
        unsigned char val = (unsigned char)((log_spiral + 1.0f) * 0.5f * 255.0f);
        data[idx + 0] = val;          // R
        data[idx + 1] = 255 - val;    // G
        data[idx + 2] = (val + frame) % 256; // B
        data[idx + 3] = 255;          // A
    }
}

__global__ void fill_matrix_mandelbrot_rgba(unsigned char* data, int width, int height, int frame, float cX, float cY, float baseScale) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x < width && y < height) {
        int idx = (y * width + x) * 4;
        double scale = baseScale * powf(0.99f, frame);
        int maxIter = 2000;
        float jx = cX + (x - width/2.0f) * scale;
        float jy = cY + (y - height/2.0f) * scale;
        float zx = 0, zy = 0;
        int iter = 0;
        while(zx*zx + zy*zy < 4.0f && iter < maxIter){
            float tmp = zx*zx - zy*zy + jx;
            zy = 2.0f*zx*zy + jy;
            zx = tmp;
            iter++;
        }
        // Normalizza iter in [0,255]
        unsigned char colorIdx = (unsigned char)((iter * 255) / maxIter);
        // Applica palette fissa
        data[idx+0] = palette_gist_ncar[colorIdx][0]; // R
        data[idx+1] = palette_gist_ncar[colorIdx][1]; // G
        data[idx+2] = palette_gist_ncar[colorIdx][2]; // B
        data[idx+3] = 255;                  // A
    }
}


class CudaSender : public QObject {
    Q_OBJECT

public:
    CudaSender() {
        ackSocket.bind(9998);

        CUDA_SAFE_CALL(hipMalloc(&d_working, size));
        CUDA_SAFE_CALL(hipMalloc(&d_framesend, size));

        CUDA_SAFE_CALL(hipIpcGetMemHandle(&memHandle, d_framesend));

        QByteArray datagram(reinterpret_cast<char*>(&memHandle), sizeof(memHandle));
        udpSocket.writeDatagram(datagram, QHostAddress("127.0.0.1"), 9999);

        connect(&timer, &QTimer::timeout, this, &CudaSender::sendFrame);
        timer.start(16); // Un fotogramma ogni 16 ms -> 60 FPS
        // timer.start(33); // Un fotogramma ogni 33 ms -> 30 FPS

        connect(&ackSocket, &QUdpSocket::readyRead, this, &CudaSender::handleIncoming);
    }

    ~CudaSender() {
        timer.stop();
        CUDA_SAFE_CALL(hipFree(d_working));
        CUDA_SAFE_CALL(hipFree(d_framesend));
    }

public slots:
    void sendFrame() {
        frame++;

        dim3 block(16, 16);
        dim3 grid((WIDTH + block.x - 1) / block.x, (HEIGHT + block.y - 1) / block.y);
        //fill_matrix_linear_rgba<<<grid, block>>>(d_working, WIDTH, HEIGHT, frame);
        //fill_matrix_spiral_rgba<<<grid, block>>>(d_working, WIDTH, HEIGHT, frame);
        fill_matrix_mandelbrot_rgba<<<grid, block>>>(d_working, WIDTH, HEIGHT, frame, CX, CY, 16.0f/WIDTH);
        CUDA_SAFE_CALL(hipGetLastError());
        CUDA_SAFE_CALL(hipDeviceSynchronize());

        #ifdef DEBUG
            unsigned char* h_debug = new unsigned char[3];
            CUDA_SAFE_CALL(hipMemcpy(h_debug, d_working, 3, hipMemcpyDeviceToHost));
            for (int i = 0; i < 3; ++i) {
                std::cout << "SENDER - h_debug[" << i << "] = " << (int)h_debug[i] << std::endl;
            }
        #endif

        CUDA_SAFE_CALL(hipMemcpy(d_framesend, d_working, size, hipMemcpyDeviceToDevice));

        udpSocket.writeDatagram("FRAME_READY", QHostAddress("127.0.0.1"), 9999);
    }

    void handleIncoming() {
        while (ackSocket.hasPendingDatagrams()) {
            QByteArray datagram;
            datagram.resize(ackSocket.pendingDatagramSize());
            ackSocket.readDatagram(datagram.data(), datagram.size());

            if (datagram == "STOP") {
                std::cout << "Received STOP. Exiting...\n";
                QCoreApplication::quit();
            }
        }
    }

private:
    int frame = 0;
    const size_t size = WIDTH * HEIGHT * CHANNELS;
    unsigned char *d_working = nullptr;
    unsigned char *d_framesend = nullptr;
    hipIpcMemHandle_t memHandle;
    QUdpSocket udpSocket, ackSocket;
    QTimer timer;
};

#include ".sender.moc"

int main(int argc, char *argv[]) {
    QCoreApplication app(argc, argv);
    CudaSender sender;
    return app.exec();
}
