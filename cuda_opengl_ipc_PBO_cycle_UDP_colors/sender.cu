#include "hip/hip_runtime.h"
#include <QCoreApplication>
#include <QUdpSocket>
#include <QTimer>
#include <QHostAddress>

#include "global_include.h"

__device__ __constant__ unsigned char palette[256][3] = {
{66,30,15},{25,7,26},{9,1,47},{4,4,73},{0,7,100},{12,44,138},{24,82,177},{57,125,209},
{134,181,229},{211,236,248},{241,233,191},{248,201,95},{255,170,0},{204,128,0},{153,87,0},{106,52,3},
{66,30,15},{25,7,26},{9,1,47},{4,4,73},{0,7,100},{12,44,138},{24,82,177},{57,125,209},
{134,181,229},{211,236,248},{241,233,191},{248,201,95},{255,170,0},{204,128,0},{153,87,0},{106,52,3},
{66,30,15},{25,7,26},{9,1,47},{4,4,73},{0,7,100},{12,44,138},{24,82,177},{57,125,209},
{134,181,229},{211,236,248},{241,233,191},{248,201,95},{255,170,0},{204,128,0},{153,87,0},{106,52,3},
{66,30,15},{25,7,26},{9,1,47},{4,4,73},{0,7,100},{12,44,138},{24,82,177},{57,125,209},
{134,181,229},{211,236,248},{241,233,191},{248,201,95},{255,170,0},{204,128,0},{153,87,0},{106,52,3},
{66,30,15},{25,7,26},{9,1,47},{4,4,73},{0,7,100},{12,44,138},{24,82,177},{57,125,209},
{134,181,229},{211,236,248},{241,233,191},{248,201,95},{255,170,0},{204,128,0},{153,87,0},{106,52,3},
{66,30,15},{25,7,26},{9,1,47},{4,4,73},{0,7,100},{12,44,138},{24,82,177},{57,125,209},
{134,181,229},{211,236,248},{241,233,191},{248,201,95},{255,170,0},{204,128,0},{153,87,0},{106,52,3},
{66,30,15},{25,7,26},{9,1,47},{4,4,73},{0,7,100},{12,44,138},{24,82,177},{57,125,209},
{134,181,229},{211,236,248},{241,233,191},{248,201,95},{255,170,0},{204,128,0},{153,87,0},{106,52,3},
{66,30,15},{25,7,26},{9,1,47},{4,4,73},{0,7,100},{12,44,138},{24,82,177},{57,125,209},
{134,181,229},{211,236,248},{241,233,191},{248,201,95},{255,170,0},{204,128,0},{153,87,0},{106,52,3},
{66,30,15},{25,7,26},{9,1,47},{4,4,73},{0,7,100},{12,44,138},{24,82,177},{57,125,209},
{134,181,229},{211,236,248},{241,233,191},{248,201,95},{255,170,0},{204,128,0},{153,87,0},{106,52,3},
{66,30,15},{25,7,26},{9,1,47},{4,4,73},{0,7,100},{12,44,138},{24,82,177},{57,125,209},
{134,181,229},{211,236,248},{241,233,191},{248,201,95},{255,170,0},{204,128,0},{153,87,0},{106,52,3},
{66,30,15},{25,7,26},{9,1,47},{4,4,73},{0,7,100},{12,44,138},{24,82,177},{57,125,209},
{134,181,229},{211,236,248},{241,233,191},{248,201,95},{255,170,0},{204,128,0},{153,87,0},{106,52,3},
{66,30,15},{25,7,26},{9,1,47},{4,4,73},{0,7,100},{12,44,138},{24,82,177},{57,125,209},
{134,181,229},{211,236,248},{241,233,191},{248,201,95},{255,170,0},{204,128,0},{153,87,0},{106,52,3},
{66,30,15},{25,7,26},{9,1,47},{4,4,73},{0,7,100},{12,44,138},{24,82,177},{57,125,209},
{134,181,229},{211,236,248},{241,233,191},{248,201,95},{255,170,0},{204,128,0},{153,87,0},{106,52,3}
};


__global__ void fill_matrix_linear_rgba(unsigned char* data, int width, int height, int frame) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x < width && y < height) {
        int idx = (y * width + x) * 4; // 4 canali
        data[idx + 0] = (x + frame) % 256; // R
        data[idx + 1] = (y + frame) % 256; // G
        data[idx + 2] = ((x + y) + frame) % 256; // B
        data[idx + 3] = 255; // A
    }
}

__global__ void fill_matrix_spiral_rgba(unsigned char* data, int width, int height, int frame) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x < width && y < height) {
        int idx = (y * width + x) * 4;

        float cx = width * 0.5f;
        float cy = height * 0.5f;
        float dx = x - cx;
        float dy = y - cy;
        float r = sqrtf(dx*dx + dy*dy);
        float theta = atan2f(dy, dx);
        float t = frame * 0.2f;
        // float lin_spiral = sinf(r * 0.05f + theta - t);
        float log_spiral = sinf(10.0f * logf(r + 1e-6f) + theta - t);
        unsigned char val = (unsigned char)((log_spiral + 1.0f) * 0.5f * 255.0f);
        data[idx + 0] = val;          // R
        data[idx + 1] = 255 - val;    // G
        data[idx + 2] = (val + frame) % 256; // B
        data[idx + 3] = 255;          // A
    }
}

__global__ void fill_matrix_mandelbrot_rgba(unsigned char* data, int width, int height, int frame, float cX, float cY, float baseScale) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x < width && y < height) {

        int idx = (y * width + x) * 4;

        double scale = baseScale * powf(0.99f, frame);
        int maxIter = 2000;

        float jx = cX + (x - width/2.0f) * scale;
        float jy = cY + (y - height/2.0f) * scale;

        float zx = 0, zy = 0;
        int iter = 0;
        while(zx*zx + zy*zy < 4.0f && iter < maxIter){
            float tmp = zx*zx - zy*zy + jx;
            zy = 2.0f*zx*zy + jy;
            zx = tmp;
            iter++;
        }

        // Normalizza iter in [0,255]
        unsigned char colorIdx = (unsigned char)((iter * 255) / maxIter);
        // Applica palette fissa
        data[idx+0] = palette[colorIdx][0]; // R
        data[idx+1] = palette[colorIdx][1]; // G
        data[idx+2] = palette[colorIdx][2]; // B
        data[idx+3] = 255;                  // A
    }
}


class CudaSender : public QObject {
    Q_OBJECT

public:
    CudaSender() {
        ackSocket.bind(9998);

        CUDA_SAFE_CALL(hipMalloc(&d_working, size));
        CUDA_SAFE_CALL(hipMalloc(&d_framesend, size));

        CUDA_SAFE_CALL(hipIpcGetMemHandle(&memHandle, d_framesend));

        QByteArray datagram(reinterpret_cast<char*>(&memHandle), sizeof(memHandle));
        udpSocket.writeDatagram(datagram, QHostAddress("127.0.0.1"), 9999);

        connect(&timer, &QTimer::timeout, this, &CudaSender::sendFrame);
        timer.start(16); // Un fotogramma ogni 16 ms -> 60 FPS
        // timer.start(33); // Un fotogramma ogni 33 ms -> 30 FPS

        connect(&ackSocket, &QUdpSocket::readyRead, this, &CudaSender::handleIncoming);
    }

    ~CudaSender() {
        timer.stop();
        CUDA_SAFE_CALL(hipFree(d_working));
        CUDA_SAFE_CALL(hipFree(d_framesend));
    }

public slots:
    void sendFrame() {
        frame++;

        dim3 block(16, 16);
        dim3 grid((WIDTH + block.x - 1) / block.x, (HEIGHT + block.y - 1) / block.y);
        //fill_matrix_linear_rgba<<<grid, block>>>(d_working, WIDTH, HEIGHT, frame);
        //fill_matrix_spiral_rgba<<<grid, block>>>(d_working, WIDTH, HEIGHT, frame);
        fill_matrix_mandelbrot_rgba<<<grid, block>>>(d_working, WIDTH, HEIGHT, frame, -0.7436438870371587f, 0.13182590420531197f, 16.0f/WIDTH);
        CUDA_SAFE_CALL(hipGetLastError());
        CUDA_SAFE_CALL(hipDeviceSynchronize());

        #ifdef DEBUG
            unsigned char* h_debug = new unsigned char[3];
            CUDA_SAFE_CALL(hipMemcpy(h_debug, d_working, 3, hipMemcpyDeviceToHost));
            for (int i = 0; i < 3; ++i) {
                std::cout << "SENDER - h_debug[" << i << "] = " << (int)h_debug[i] << std::endl;
            }
        #endif

        CUDA_SAFE_CALL(hipMemcpy(d_framesend, d_working, size, hipMemcpyDeviceToDevice));

        udpSocket.writeDatagram("FRAME_READY", QHostAddress("127.0.0.1"), 9999);
    }

    void handleIncoming() {
        while (ackSocket.hasPendingDatagrams()) {
            QByteArray datagram;
            datagram.resize(ackSocket.pendingDatagramSize());
            ackSocket.readDatagram(datagram.data(), datagram.size());

            if (datagram == "STOP") {
                std::cout << "Received STOP. Exiting...\n";
                QCoreApplication::quit();
            }
        }
    }

private:
    int frame = 0;
    const size_t size = WIDTH * HEIGHT * CHANNELS;
    unsigned char *d_working = nullptr;
    unsigned char *d_framesend = nullptr;
    hipIpcMemHandle_t memHandle;
    QUdpSocket udpSocket, ackSocket;
    QTimer timer;
};

#include ".sender.moc"

int main(int argc, char *argv[]) {
    QCoreApplication app(argc, argv);
    CudaSender sender;
    return app.exec();
}
