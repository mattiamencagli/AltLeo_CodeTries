#include "hip/hip_runtime.h"
#include <iostream>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipcub/hipcub.hpp>

#define USE_NVTX
#ifdef USE_NVTX
#include "nvtx3/nvToolsExt.h"

// ARGB hexadecimal notation. 
// Always put alpha channel at max to properly visualize the colors in Nsight,
// thus, each color must start with "ff"
#define GREEN   0xff00ff00
#define BLUE    0xff0000ff
#define YELLOW  0xffffff00
#define MAGENTA 0xffff00ff
#define CYAN    0xff00ffff
#define RED     0xffff0000
#define SILVER  0xffc0c0c0

#define NVTX_START(name,cid) { \
    nvtxEventAttributes_t eventAttrib = {0}; \
    eventAttrib.version = NVTX_VERSION; \
    eventAttrib.size = NVTX_EVENT_ATTRIB_STRUCT_SIZE; \
    eventAttrib.colorType = NVTX_COLOR_ARGB; \
    eventAttrib.color = cid; \
    eventAttrib.messageType = NVTX_MESSAGE_TYPE_ASCII; \
    eventAttrib.message.ascii = name; \
    nvtxRangePushEx(&eventAttrib); \
}
#define NVTX_STOP(name) nvtxRangePop();
#else
#define NVTX_START(name,cid)
#define NVTX_STOP(name)
#endif

#define DEBUG
#ifdef DEBUG
#define CUDA_SAFE_CALL(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line) {
    if (code != hipSuccess) {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        exit(code);
    }
}
#else
#define CUDA_SAFE_CALL(ans)
#endif


#define N_THREADS 512
#define N_BLOCKS 24
static const int blockSize = 512;
static const int gridSize = 24;

__device__ bool lastBlockMaxAbs(unsigned int* counter) {
    __threadfence();
    int last = 0;
    if(threadIdx.x == 0)
        last = atomicAdd(counter,1);
    return __syncthreads_or(last == gridDim.x-1);
}

__global__ void localmaxabs(float *input, const int arraySize, float *output, unsigned int* lastBlockCounter) {
    int thIdx = threadIdx.x;
    int gthIdx = thIdx + blockIdx.x*blockSize;
    const int gridSize = blockSize*gridDim.x;
    float value;

    float valuemax = 0;

    int h = gthIdx - (gthIdx/1280)*1280;
    int k = gthIdx/1280;

    for(int i = gthIdx; i < arraySize; i += gridSize)
    {
        value = fabs(input[i]);
        if(value >  valuemax  && h>=4  &&h<1276 && k>=4 && k<716) { //Esclusione dei bordi dal calcolo del max
            valuemax = value;
        }
        output[i] = 0.0f;
    }

    __shared__  float shArr[blockSize];
    shArr[thIdx] = valuemax;
    __syncthreads();
    for(int size = blockSize/2; size>0; size/=2) {
        if(thIdx<size)
        {
            float value1 = shArr[thIdx];
            float value2 = shArr[thIdx+size];
            if(value2 > value1)
            {
                shArr[thIdx]=value2;
            }
        }
        __syncthreads();
    }
    if(thIdx == 0)
        output[blockIdx.x] = shArr[0];
    if(lastBlockMaxAbs(lastBlockCounter)) {
        shArr[thIdx] = thIdx<gridSize ? output[thIdx] : 0;
        __syncthreads();
        for(int size = blockSize/2; size>0; size/=2) {
            if(thIdx<size)
            {
                float value1 = shArr[thIdx];
                float value2 = shArr[thIdx+size];
                if(value2 > value1)
                {
                    shArr[thIdx]=value2;
                }

            }
            __syncthreads();
        }
        if(thIdx == 0)
            output[0] = shArr[0];
    }
}



int main(){

	NVTX_START("ALL", GREEN);

	NVTX_START("Init", RED);
	int deviceId, numberOfSMs;
	CUDA_SAFE_CALL(hipGetDevice(&deviceId));
	CUDA_SAFE_CALL(hipDeviceGetAttribute(&numberOfSMs, hipDeviceAttributeMultiprocessorCount, deviceId));
	printf("device ID = %d; Number of SMs = %d\n", deviceId, numberOfSMs);

	hipEvent_t start, stop;
	CUDA_SAFE_CALL(hipEventCreate(&start));
	CUDA_SAFE_CALL(hipEventCreate(&stop));

	//hipStream_t cudaStream;
	//CUDA_SAFE_CALL(hipStreamCreate(&cudaStream));

	const int nx = 1280;
	const int ny = 720;
	const int N  = nx*ny;
	NVTX_STOP("Init");

	NVTX_START("Allocations", BLUE);
	float * im = (float*) malloc(N*sizeof(float));
	memset(im, 0, N*sizeof(float));
	im[1401]   = 7.0f;
	im[2008]   = 9.0f;
	im[3402]   = 11.0f;
	im[7007]   = 13.0f;
	im[9999]   = 15.0f;
	im[12345]  = 17.0f;
	im[98765]  = 19.0f;
	im[123456] = 21.0f;
	im[500500] = 23.0f;
	im[920123] = 42.0f;


	NVTX_START("Copy array to GPU", MAGENTA);
	float * im_gpu;
	CUDA_SAFE_CALL(hipMalloc<float>(&im_gpu, N*sizeof(float)));
	CUDA_SAFE_CALL(hipMemcpy(im_gpu, im, N*sizeof(float), hipMemcpyHostToDevice));
	NVTX_STOP("Copy array to GPU")

	float * MAX_gpu;
	CUDA_SAFE_CALL(hipMalloc<float>(&MAX_gpu, sizeof(float)));

	unsigned int * lastblockcounter;
   	CUDA_SAFE_CALL(hipMalloc(&lastblockcounter, sizeof(unsigned int)));
     	CUDA_SAFE_CALL(hipMemset(lastblockcounter,0,sizeof(unsigned int)));
	NVTX_STOP("Allocations");
	
	NVTX_START("Kernel", SILVER);
	CUDA_SAFE_CALL(hipEventRecord(start, 0));
	CUDA_SAFE_CALL(hipEventSynchronize(start));
	localmaxabs<<<gridSize,blockSize>>>(im_gpu, N, MAX_gpu, lastblockcounter);
        CUDA_SAFE_CALL(hipGetLastError());
	CUDA_SAFE_CALL(hipEventRecord(stop, 0));
	CUDA_SAFE_CALL(hipEventSynchronize(stop));
	float time_ms;
	CUDA_SAFE_CALL(hipEventElapsedTime(&time_ms, start, stop));
	NVTX_STOP("Kernel");

	NVTX_START("Check max", YELLOW);
	float MAX = 12345.0f;
	CUDA_SAFE_CALL(hipMemcpy(&MAX, MAX_gpu, sizeof(float), hipMemcpyDeviceToHost));
	CUDA_SAFE_CALL(hipDeviceSynchronize());
	NVTX_STOP("Check max");
	
	printf("MAX = %f (expected=42);   total time = %f ms\n",MAX, time_ms);
	
	NVTX_START("Finalize", CYAN);
	CUDA_SAFE_CALL(hipEventDestroy(start));
	CUDA_SAFE_CALL(hipEventDestroy(stop));
	//CUDA_SAFE_CALL(hipStreamDestroy(&cudaStream));
	NVTX_STOP("Finalize");
	
	NVTX_STOP("ALL");

}

