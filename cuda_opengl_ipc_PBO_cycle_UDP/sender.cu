#include "hip/hip_runtime.h"
#include <QCoreApplication>
#include <QUdpSocket>
#include <QTimer>
#include <QHostAddress>

#include "global_include.h"

__global__ void fill_matrix_linear(unsigned char* data, int width, int height, int frame) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x < width && y < height) {
        int idx = y * width + x;
        data[idx] = (x + y + frame) % 256;
    }
}

__global__ void fill_matrix_spiral(unsigned char* data, int width, int height, int frame) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x < width && y < height) {
        int idx = y * width + x;
        float cx = width * 0.5f;
        float cy = height * 0.5f;
        float dx = x - cx;
        float dy = y - cy;
        float r = sqrtf(dx * dx + dy * dy);
        float theta = atan2f(dy, dx);
        float t = frame * 0.2f;
        // float spiral = sinf(r * 0.05f + theta - t);
        float spiral = sinf( 10.0f * logf(r + 1e-6f) + theta - t);
        int val = int((spiral + 1.0f) * 0.5f * 255.0f);
        data[idx] = (unsigned char)(val % 256);
    }
}

class CudaSender : public QObject {
    Q_OBJECT

public:
    CudaSender() {
        ackSocket.bind(9998);

        CUDA_SAFE_CALL(hipMalloc(&d_working, size));
        CUDA_SAFE_CALL(hipMalloc(&d_framesend, size));

        CUDA_SAFE_CALL(hipIpcGetMemHandle(&memHandle, d_framesend));

        QByteArray datagram(reinterpret_cast<char*>(&memHandle), sizeof(memHandle));
        udpSocket.writeDatagram(datagram, QHostAddress("127.0.0.1"), 9999);

        connect(&timer, &QTimer::timeout, this, &CudaSender::sendFrame);
        timer.start(33);

        connect(&ackSocket, &QUdpSocket::readyRead, this, &CudaSender::handleIncoming);
    }

    ~CudaSender() {
        timer.stop();
        CUDA_SAFE_CALL(hipFree(d_working));
        CUDA_SAFE_CALL(hipFree(d_framesend));
    }

public slots:
    void sendFrame() {
        frame++;

        dim3 block(16, 16);
        dim3 grid((WIDTH + block.x - 1) / block.x, (HEIGHT + block.y - 1) / block.y);
        fill_matrix_spiral<<<grid, block>>>(d_working, WIDTH, HEIGHT, frame);
        CUDA_SAFE_CALL(hipGetLastError());
        CUDA_SAFE_CALL(hipDeviceSynchronize());

        #ifdef DEBUG
            unsigned char* h_debug = new unsigned char[10];
            CUDA_SAFE_CALL(hipMemcpy(h_debug, d_working, 10, hipMemcpyDeviceToHost));
            for (int i = 0; i < 3; ++i) {
                std::cout << "SENDER - h_debug[" << i << "] = " << (int)h_debug[i] << std::endl;
            }
        #endif

        CUDA_SAFE_CALL(hipMemcpy(d_framesend, d_working, size, hipMemcpyDeviceToDevice));

        udpSocket.writeDatagram("FRAME_READY", QHostAddress("127.0.0.1"), 9999);
    }

    void handleIncoming() {
        while (ackSocket.hasPendingDatagrams()) {
            QByteArray datagram;
            datagram.resize(ackSocket.pendingDatagramSize());
            ackSocket.readDatagram(datagram.data(), datagram.size());

            if (datagram == "STOP") {
                std::cout << "Received STOP. Exiting...\n";
                QCoreApplication::quit();
            }
        }
    }

private:
    int frame = 0;
    const size_t size = WIDTH * HEIGHT;
    unsigned char *d_working = nullptr;
    unsigned char *d_framesend = nullptr;
    hipIpcMemHandle_t memHandle;
    QUdpSocket udpSocket, ackSocket;
    QTimer timer;
};

#include ".sender.moc"

int main(int argc, char *argv[]) {
    QCoreApplication app(argc, argv);
    CudaSender sender;
    return app.exec();
}
