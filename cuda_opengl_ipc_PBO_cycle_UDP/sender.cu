#include "hip/hip_runtime.h"
#include <QCoreApplication>
#include <QUdpSocket>
#include <QTimer>
#include <QHostAddress>

#include "global_include.h"

__global__ void fill_matrix(unsigned char* data, int width, int height, int frame, int speed) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x < width && y < height) {
        int idx = y * width + x;
        data[idx] = (x + y + frame*speed) % 256;
    }
}

class CudaSender : public QObject {
    Q_OBJECT

public:
    CudaSender() {
        ackSocket.bind(9998);

        CUDA_SAFE_CALL(hipMalloc(&d_working, size));
        CUDA_SAFE_CALL(hipMalloc(&d_framesend, size));

        CUDA_SAFE_CALL(hipIpcGetMemHandle(&memHandle, d_framesend));

        QByteArray datagram(reinterpret_cast<char*>(&memHandle), sizeof(memHandle));
        udpSocket.writeDatagram(datagram, QHostAddress("127.0.0.1"), 9999);

        connect(&timer, &QTimer::timeout, this, &CudaSender::sendFrame);
        timer.start(33);

        connect(&ackSocket, &QUdpSocket::readyRead, this, &CudaSender::handleIncoming);
    }

    ~CudaSender() {
        timer.stop();
        CUDA_SAFE_CALL(hipFree(d_working));
        CUDA_SAFE_CALL(hipFree(d_framesend));
    }

public slots:
    void sendFrame() {
        frame++;

        dim3 block(16, 16);
        dim3 grid((WIDTH + block.x - 1) / block.x, (HEIGHT + block.y - 1) / block.y);
        fill_matrix<<<grid, block>>>(d_working, WIDTH, HEIGHT, frame, 4);
        CUDA_SAFE_CALL(hipGetLastError());
        CUDA_SAFE_CALL(hipDeviceSynchronize());

        #ifdef DEBUG
            unsigned char* h_debug = new unsigned char[10];
            hipMemcpy(h_debug, d_working, 10, hipMemcpyDeviceToHost);
            for (int i = 0; i < 3; ++i) {
                std::cout << "SENDER - h_debug[" << i << "] = " << (int)h_debug[i] << std::endl;
            }
        #endif

        CUDA_SAFE_CALL(hipMemcpy(d_framesend, d_working, size, hipMemcpyDeviceToDevice));

        udpSocket.writeDatagram("FRAME_READY", QHostAddress("127.0.0.1"), 9999);
    }

    void handleIncoming() {
        while (ackSocket.hasPendingDatagrams()) {
            QByteArray datagram;
            datagram.resize(ackSocket.pendingDatagramSize());
            ackSocket.readDatagram(datagram.data(), datagram.size());

            if (datagram == "STOP") {
                std::cout << "Received STOP. Exiting...\n";
                QCoreApplication::quit();
            }
        }
    }

private:
    int frame = 0;
    const size_t size = WIDTH * HEIGHT;
    unsigned char *d_working = nullptr;
    unsigned char *d_framesend = nullptr;
    hipIpcMemHandle_t memHandle;
    QUdpSocket udpSocket, ackSocket;
    QTimer timer;
};

#include ".sender.moc"

int main(int argc, char *argv[]) {
    QCoreApplication app(argc, argv);
    CudaSender sender;
    return app.exec();
}
